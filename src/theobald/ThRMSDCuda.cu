#include "ThRMSDCuda.cuh"
#include "kernel_functions_cuda.cuh"
#include <iostream>
using namespace std;

#define floating_point_type float

inline void checkCudaError(char* id, hipError_t error_code){
	if (error_code != 0){
		cout<<"Error in "<<id<<" . Error code: "<<error_code<<". Exiting..."<<flush<<endl;
		exit(-1);
	}
}

void ThRMSDCuda::cudaInit(){
	hipDeviceProp_t props;
	int device;
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device);
	
	if(!props.deviceOverlap){
		cout<<"No multiple streams"<<endl;
	}
}

ThRMSDCuda::ThRMSDCuda(int numberOfConformations, int atomsPerConformation, double* coords, int threads_per_block, int number_of_blocks):
				RMSD(numberOfConformations, atomsPerConformation, coords){
    
    cudaInit();
    
    
	int total_num_of_coords = this->numberOfConformations*this->atomsPerConformation*3;
    
    // Convert to our floating point type
    this->tmpHostCoords = new floating_point_type[total_num_of_coords];
    for(int i = 0; i < total_num_of_coords; ++i){
    	this->tmpHostCoords[i] = (floating_point_type) coords[i];
    }
    
	// Allocate space for temporary rmsds
	this->tmpHostRMSDs = new floating_point_type[numberOfConformations];
    
	//////////////////////////////////////////////////////////
	//// Set up GPU
	//////////////////////////////////////////////////////////
	this->threadsPerBlock = threads_per_block;
	this->numberOfBlocks = number_of_blocks;
	
	
	// GPU Data allocation for input
	checkCudaError(" Malloc Device Coords. ", hipMalloc((void **) &deviceCoords, total_num_of_coords * sizeof(floating_point_type)));
	checkCudaError(" Copying Coords to Device. ", hipMemcpy(deviceCoords, tmpHostCoords, total_num_of_coords * sizeof(floating_point_type), 
			hipMemcpyHostToDevice));
    
    // GPU Data allocation for output
    checkCudaError(" Malloc RMSDs ", hipMalloc((void **) &deviceRMSDs, numberOfConformations * sizeof(floating_point_type)));
   
    
    
    //////////////////////////////////////////////////////////
	//// And do some pre-processing
	//////////////////////////////////////////////////////////
    centerCoordsOfAllConformations<<<numberOfBlocks, threadsPerBlock>>>(numberOfConformations,atomsPerConformation,deviceCoords);
}

ThRMSDCuda::~ThRMSDCuda(){
    checkCudaError(" deviceCoords hipFree ", hipFree(this->deviceCoords));
    checkCudaError(" deviceRMSDs hipFree ", hipFree(this->deviceRMSDs));
    delete [] this->tmpHostRMSDs;
    delete [] this->tmpHostCoords;
}

void ThRMSDCuda::oneVsTheOthers(int conformation, double* rmsd_result) {
	if (conformation < numberOfConformations){
	    calcRMSDOfOneVsOthers<<<numberOfBlocks,threadsPerBlock>>>(this->deviceCoords, conformation, conformation + 1, 
	    															numberOfConformations, atomsPerConformation, 
	    															atomsPerConformation*3, this->deviceRMSDs);
	    
	    checkCudaError(" Getting RMSDs from Device. ", hipMemcpy(this->tmpHostRMSDs, this->deviceRMSDs, 
	    											   numberOfConformations * sizeof(floating_point_type), 
	    											   hipMemcpyDeviceToHost));
	    
	    // Do the copy to the output vector (need to have the correct size)
		int j = 0;
		for (int i = conformation + 1; i < numberOfConformations;++i,++j){
			rmsd_result[j] = (double) this->tmpHostRMSDs[i];
		}
	}
}

void ThRMSDCuda::calculateRMSDCondensedMatrix(vector<double>& rmsd){ 
	
	hipStream_t copy_stream, execution_stream;
	hipStreamCreate(&copy_stream);
	hipStreamCreate(&execution_stream);
	
	this->rmsdMatrixLen = (numberOfConformations*(numberOfConformations-1))/2;
	checkCudaError("tmpHostRMSD hipHostAlloc ", hipHostAlloc((void**)&(this->tmpHostRMSDMatrix), 
												rmsdMatrixLen*sizeof(floating_point_type),hipHostMallocDefault));
	
	float time;
	hipEvent_t start;
	hipEventCreate(&start);
	hipEvent_t end;
	hipEventCreate( &end);

	
		hipEventRecord(start, 0);
		int rmsdMatrixOffset = 0;
		int numberOfCalculatedRmsds = 0;
		for(int conformation_number = 0;conformation_number < numberOfConformations; ++conformation_number){ 
			numberOfCalculatedRmsds = numberOfConformations-(conformation_number+1);
	    	
	    	calcRMSDOfOneVsOthers<<<numberOfBlocks,
	    						threadsPerBlock,
	    						0,execution_stream>>>(deviceCoords, conformation_number, conformation_number + 1, 
	    																					numberOfConformations, atomsPerConformation, 
	    																					atomsPerConformation*3, deviceRMSDs);
			hipStreamSynchronize(execution_stream);
		
	    	checkCudaError(" Getting RMSDs from Device. ", hipMemcpyAsync(	&(tmpHostRMSDMatrix[rmsdMatrixOffset]), 
	    																&(deviceRMSDs[conformation_number+1]), 
	    											   					numberOfCalculatedRmsds * sizeof(floating_point_type), 
	    											   					hipMemcpyDeviceToHost,
	    											   					copy_stream));
	    	hipStreamSynchronize(copy_stream);
	    	
			rmsdMatrixOffset += numberOfCalculatedRmsds;
		}
		
		for (int i = 0; i < rmsdMatrixLen; ++i){
			rmsd.push_back((double)this->tmpHostRMSDMatrix[i]);
		}
		
		hipEventRecord(end, 0 );
	    hipEventSynchronize(end);
	    
	    hipEventElapsedTime(&time, start, end);	
	    cout<<"Time for calculations (ms):"<<time<<endl;	
	     
    hipEventDestroy(start);
    hipEventDestroy(end);
    
    hipStreamDestroy(copy_stream);
	hipStreamDestroy(execution_stream);
    checkCudaError("tmpHostRMSD hipHostFree ",hipHostFree(tmpHostRMSDMatrix));
}

void ThRMSDCuda::getDeviceCoordinates(vector<double>& coordinates){
	int number_of_coords = numberOfConformations*atomsPerConformation*3;
    floating_point_type* tmpCoords = new floating_point_type[number_of_coords];
	checkCudaError(" Getting Coordinates from Device. ", 
		hipMemcpy(tmpCoords, deviceCoords, number_of_coords * sizeof(floating_point_type), hipMemcpyDeviceToHost)
	);
	coordinates.clear();
	for (int i = 0; i < number_of_coords; ++i){
		coordinates.push_back((double)tmpCoords[i]);
	}
	delete [] tmpCoords;
}