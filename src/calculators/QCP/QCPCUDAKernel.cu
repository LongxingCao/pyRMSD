#include <cmath>
#include <iostream>
#include "kernel_functions_cuda.h"
#include "QCPCUDAKernel.h"
#include "../RMSDTools.h"
using namespace std;

/**
 *	Convenience function for CUDA error handling. If it captures an error, writes a user-specified
 *  message, and exits the program.
 *
 *  \param 	message [In] Message to print if something went wrong (usually calling CUDA function name).
 *  \param 	error_code [In] CUDA error code (It must be 0 if everything is OK).
 */
inline void checkCudaError(char* message, hipError_t error_code){
	if (error_code != 0){
		std::cout<<"Error in "<<message<<". Error code: "<<error_code<<". Exiting..."<<std::flush<<std::endl;
		exit(-1);
	}
}

/**
 * Kernel creator.
 */
QCPCUDAKernel::QCPCUDAKernel(
				double* coordinates, 
				int atomsPerConformation,
				int coordinatesPerConformation, 
				int numberOfConformations,
				int threads_per_block, 
				int blocks_per_grid){
	
	//	Note: I'm  not using streams anymore as it didn't give me any speedup
	this->threads_per_block = threads_per_block;
	this->blocks_per_grid = blocks_per_grid;
	
	tmpHostCoords = tmpHostRMSDs = tmpCalcHostCoords = NULL;
	
	deviceCoords =  deviceRMSDs = deviceReference =
	deviceCalcCoords = deviceCalcReference = NULL;

	int totalNumberOfCoordinates = coordinatesPerConformation*numberOfConformations;

	#ifdef CUDA_PRECISION_SINGLE
		// Allocate space for buffers
		this->tmpHostCoords = new float[totalNumberOfCoordinates];
		this->tmpHostRMSDs = new float[numberOfConformations];
	#endif
		
	// GPU Data allocation for input
	checkCudaError("Malloc Device Coords", 
				hipMalloc(
						(void **) &deviceReference, 
						coordinatesPerConformation * sizeof(floating_point_type)));
	
	checkCudaError("Malloc Device Coords", 
			hipMalloc(
					(void **) &deviceCoords, 
					totalNumberOfCoordinates * sizeof(floating_point_type)));
	

	// GPU Data allocation for output
	checkCudaError("Malloc RMSDs", 
			hipMalloc(
					(void **) &deviceRMSDs, 
					numberOfConformations * sizeof(floating_point_type)));
}

void QCPCUDAKernel::setCalculationCoords(
		double* calcCoords,
		int number_of_atoms,
		int numberOfConformations){

	#ifdef CUDA_PRECISION_SINGLE
			// Allocate space for temporary coords and copy contents
			this->tmpCalcHostCoords = new float[numberOfConformations*number_of_atoms*3];
	#endif

	checkCudaError("Malloc Device Calc Reference",
				hipMalloc(
						(void **) &deviceCalcReference,
						number_of_atoms * 3 * sizeof(floating_point_type)));

	checkCudaError("Malloc Device Calc Coords",
			hipMalloc(
					(void **) &deviceCalcCoords,
					number_of_atoms * 3 * numberOfConformations * sizeof(floating_point_type)));
}

QCPCUDAKernel::~QCPCUDAKernel(){
	checkCudaError("deviceReference hipFree",
				hipFree(this->deviceReference));
	checkCudaError("deviceCoords hipFree", 
			hipFree(this->deviceCoords));
	checkCudaError("deviceRMSDs hipFree", 
			hipFree(this->deviceRMSDs));
	if (tmpHostCoords != NULL){
		delete [] tmpHostCoords;
	}
	if (tmpHostRMSDs != NULL){
		delete [] tmpHostRMSDs;
	}
	if (tmpCalcHostCoords != NULL){
		delete [] tmpCalcHostCoords;
	}
	if (deviceCalcCoords != NULL){
		checkCudaError("deviceCalcCoords hipFree", 
				hipFree(this->deviceCalcCoords));
	}
	if (deviceCalcReference != NULL){
		checkCudaError("deviceCalcReference hipFree", 
				hipFree(this->deviceCalcReference));
	}
}


void QCPCUDAKernel::updateDeviceCoordinates(
		double * coordinates,
		floating_point_type* device_coordinates,
		float* tmp_coordinates,
		int coordinates_per_conformation,
		int number_of_conformations){
	
	int total_number_of_coordinates = coordinates_per_conformation*number_of_conformations;
	
	#ifdef CUDA_PRECISION_SINGLE
		// Convert to our floating point type
		for(int i = 0; i < total_number_of_coordinates; ++i){
			tmp_coordinates[i] = static_cast<float>( coordinates[i] );
		}
		
		checkCudaError("Copying Coords to Device (single)", 
					hipMemcpy(
							device_coordinates, 
							tmp_coordinates, 
							total_number_of_coordinates * sizeof(float),
							hipMemcpyHostToDevice));
	#else	
		checkCudaError("Copying Coords to Device (double)", 
				hipMemcpy(
						device_coordinates, 
						coordinates, 
						total_number_of_coordinates * sizeof(double),
						hipMemcpyHostToDevice));
	#endif
}

void QCPCUDAKernel::updateHostCoordinates(
		double * coordinates,
		floating_point_type * device_coordinates,
		float * tmp_coordinates,
		int number_of_conformations,
		int coordinates_per_conformation
		){
	
	int total_number_of_coordinates = coordinates_per_conformation*number_of_conformations;
	
	#ifdef CUDA_PRECISION_SINGLE
		// Convert to our floating point type
		checkCudaError("Copying Coords to Host (single)", 
				hipMemcpy(
						tmp_coordinates, 
						device_coordinates,
						total_number_of_coordinates * sizeof(float),
						hipMemcpyDeviceToHost));
		
		// Conversion
		for(int i = 0; i < total_number_of_coordinates; ++i){
			coordinates[i] = static_cast<double>( tmp_coordinates[i]);
		}
		
	#else	
		checkCudaError("Copying Coords to Host (single)", 
				hipMemcpy(
						coordinates, 
						device_coordinates, 
						total_number_of_coordinates * sizeof(double),
						hipMemcpyDeviceToHost));
	#endif
}

void QCPCUDAKernel::updateHostRMSDs(
				int numberOfConformations,
				int reference_conformation_number,
				double* rmsd){
	
	#ifdef CUDA_PRECISION_SINGLE
		// Get RMSDs from device
		checkCudaError("Getting RMSDs from Device (single)", 
				hipMemcpy(
						this->tmpHostRMSDs,
						this->deviceRMSDs,
						(numberOfConformations - reference_conformation_number-1) * sizeof(float),
						hipMemcpyDeviceToHost));
		
		// Apply conversion
		for (int i = 0 ; i < numberOfConformations - reference_conformation_number-1;++i){
			rmsd[i] = (double) this->tmpHostRMSDs[i];
		}
		
	#else	
		checkCudaError("Getting RMSDs from Device (double)", 
				hipMemcpy(
						rmsd,
						this->deviceRMSDs,
						(numberOfConformations - reference_conformation_number-1) * sizeof(double),
						hipMemcpyDeviceToHost));
	#endif

}


void QCPCUDAKernel::oneVsFollowingFitEqualCalcCoords(
		double* reference,
		int reference_conformation_number,
		double* rmsd,
		int numberOfConformations,
		int coordinatesPerConformation,
		int atomsPerConformation,
		double *allCoordinates){

	// Update reference in device
	updateDeviceCoordinates(
			reference,
			deviceReference,
			tmpHostCoords,
			coordinatesPerConformation,
			1);
	
	// Put the centered coordinates on the device
	updateDeviceCoordinates(
			allCoordinates,
			deviceCoords,
			tmpHostCoords,
			coordinatesPerConformation,
			numberOfConformations);

	// Do the calculations
	calcRMSDOfOneVsFollowingWithRotation CUDA_KERNEL_DIM(this->blocks_per_grid, this->threads_per_block)(
			this->deviceReference, 
			reference_conformation_number,
			this->deviceCoords,
			numberOfConformations,
			atomsPerConformation,
			coordinatesPerConformation,
			this->deviceRMSDs);

	updateHostCoordinates(
			allCoordinates,
			deviceCoords,
			tmpHostCoords,
			numberOfConformations,
			coordinatesPerConformation);
	
	if(rmsd!=NULL){
		// Get RMSDs
		updateHostRMSDs(
					numberOfConformations,
					reference_conformation_number,
					rmsd);
	}
}


void QCPCUDAKernel::oneVsFollowingFitDiffersCalcCoords(
		double* fitReference,
		double* calcReference,
		int reference_conformation_number,
		double* rmsd,
		int numberOfConformations,
		int coordinatesPerConformation,
		int atomsPerConformation,
		double *allCoordinates,
		int coordinatesPerRMSDConformation,
		int atomsPerRMSDConformation,
		double *allRMSDCoordinates){
	// Update references in device
	updateDeviceCoordinates(
			fitReference,
			deviceReference,
			tmpHostCoords,
			coordinatesPerConformation,
			1);
	
	// Update the (already centered) coordinates on the device
	updateDeviceCoordinates(
			allCoordinates,
			deviceCoords,
			tmpHostCoords,
			coordinatesPerConformation,
			numberOfConformations);
	
	if(calcReference!=NULL){
		updateDeviceCoordinates(
				calcReference,
				deviceCalcReference,
				tmpCalcHostCoords,
				coordinatesPerRMSDConformation,
				1);
	}
	
	updateDeviceCoordinates(
			allRMSDCoordinates,
			deviceCalcCoords,
			tmpCalcHostCoords,
			coordinatesPerRMSDConformation,
			numberOfConformations);

	// Do the calculations
	calcRMSDOfOneVsFollowingFitDiffersCalc CUDA_KERNEL_DIM(this->blocks_per_grid, this->threads_per_block)(
			deviceReference,
			deviceCalcReference,
			reference_conformation_number,
			deviceRMSDs,
			numberOfConformations,
			coordinatesPerConformation,
			atomsPerConformation,
			deviceCoords,
			coordinatesPerRMSDConformation,
			atomsPerRMSDConformation,
			deviceCalcCoords);

	updateHostCoordinates(
				allCoordinates,
				deviceCoords,
				tmpHostCoords,
				numberOfConformations,
				coordinatesPerConformation);
	
	updateHostCoordinates(
				allRMSDCoordinates,
				deviceCalcCoords,
				tmpCalcHostCoords,
				numberOfConformations,
				coordinatesPerRMSDConformation);
	
	if(rmsd!=NULL){
		// Get RMSDs
		updateHostRMSDs(
					numberOfConformations,
					reference_conformation_number,
					rmsd);
	}

}


void QCPCUDAKernel::matrixOneVsFollowingFitEqualCalc(
									double* reference, 
									int reference_conformation_number, 
									double* rmsd,
									int numberOfConformations, 
									int coordinatesPerConformation,
									int atomsPerConformation, 
									double* allCoordinates){
	
	floating_point_type* tmpDeviceReference = &(this->deviceCoords[reference_conformation_number*coordinatesPerConformation]);
	
	// Do the calculations
	calcRMSDOfOneVsFollowing CUDA_KERNEL_DIM(this->blocks_per_grid, this->threads_per_block)(
			tmpDeviceReference, 
			reference_conformation_number,
			this->deviceCoords,
			numberOfConformations,
			atomsPerConformation,
			coordinatesPerConformation,
			this->deviceRMSDs);

	// Get RMSDs
	updateHostRMSDs(
			numberOfConformations,
			reference_conformation_number,
			rmsd);
}

void QCPCUDAKernel::matrixOneVsFollowingFitDiffersCalc(
											double* fitReference, 
											double* calcReference,
											int reference_conformation_number, 
											double* rmsd,
											int numberOfConformations, 
											int coordinatesPerConformation,
											int atomsPerConformation, 
											double* allCoordinates,
											int coordinatesPerRMSDConformation, 
											int atomsPerRMSDConformation,
											double* allRMSDCoordinates){
		
	floating_point_type* tmpFitDeviceReference = &(this->deviceCoords[reference_conformation_number*coordinatesPerConformation]);
	floating_point_type* tmpCalcDeviceReference = &(this->deviceCalcCoords[reference_conformation_number*coordinatesPerRMSDConformation]);
	
	// Do the calculations
	calcRMSDOfOneVsFollowingFitDiffersCalc CUDA_KERNEL_DIM(this->blocks_per_grid, this->threads_per_block)(
			tmpFitDeviceReference,
			tmpCalcDeviceReference,
			reference_conformation_number,
			deviceRMSDs,
			numberOfConformations,
			coordinatesPerConformation,
			atomsPerConformation,
			deviceCoords,
			coordinatesPerRMSDConformation,
			atomsPerRMSDConformation,
			deviceCalcCoords);

	updateHostRMSDs(
				numberOfConformations,
				reference_conformation_number,
				rmsd);
					
}

void QCPCUDAKernel::matrixInit(
						double* allFittingCoordinates,
						int coordinatesPerFittingConformation,
						double* allCalculationCoordinates,
						int coordinatesPerCalculationConformation,
						int numberOfConformations){

	// Put the centered coordinates on the device
	updateDeviceCoordinates(
			allFittingCoordinates,
			deviceCoords,
			tmpHostCoords,
			coordinatesPerFittingConformation,
			numberOfConformations);

	// And if we have the others, then update them
	if(allCalculationCoordinates != NULL){
		updateDeviceCoordinates(
				allCalculationCoordinates,
				deviceCalcCoords,
				tmpCalcHostCoords,
				coordinatesPerCalculationConformation,
				numberOfConformations);
	}
}
